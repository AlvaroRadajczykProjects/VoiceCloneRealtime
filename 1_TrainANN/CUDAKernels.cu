#include "hip/hip_runtime.h"
#include "CUDAKernels.cuh"

using namespace std;

void edu_shuffle(int arr[], int n) {
    unsigned semilla = rand() % 10000;
    shuffle(arr, arr + n, default_random_engine(semilla));
}

void manageCUDAError(hipError_t status, char* description) {
    if (status != hipSuccess) {
        fprintf(stderr, "\n\nError de CUDA %s: %s\n\n", description, hipGetErrorString(status));
        exit(EXIT_FAILURE);
    }
}

unsigned long long nextMultiple(unsigned long long val, int mod) {
    if (val == 0) { return mod; }
    if (val % mod == 0) { return val; }
    return val + (mod - (val % mod));
}

unsigned long long nextFourMultiple(unsigned long long val) {
    if (val == 0) { return 4; }
    if (val % 4 == 0) { return val; }
    return val + (4 - (val % 4));
}

const void matrizTraspuestaDevice(hipblasHandle_t handle, float* odata, float* idata, int m, int n) {
    hipblasSgeam_64(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, &alpha, idata, n, &beta_nosum, idata, m, odata, m);
}

const void productoMatricesDevice(hipblasHandle_t handle, const float* a, const float* b, float* c, int m, int k, int n) {
    hipblasSgemm_64(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, b, n, a, k, &beta_nosum, c, n);
}

const void productoMatricesTrasposedBDevice(hipblasHandle_t handle, const float* a, const float* b, float* c, int m, int k, int n) {
    hipblasSgemm_64(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k, &alpha, b, k, a, k, &beta_nosum, c, n);
}

const void productoMatricesTrasposedADevice(hipblasHandle_t handle, const float* a, const float* b, float* c, int m, int k, int n) {
    hipblasSgemm_64(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, m, k, &alpha, b, n, a, m, &beta_nosum, c, n);
}

const void productoMatricesBatchDevice(hipblasHandle_t handle, float** a, float** b, float** c, int m, int k, int n, int num_matr) {
    hipblasSgemmBatched_64(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, b, n, a, k, &beta_nosum, c, n, num_matr);
}

const void productoMatricesTrasposedBBatchDevice(hipblasHandle_t handle, float** a, float** b, float** c, int m, int k, int n, int num_matr) {
    hipblasSgemmBatched_64(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k, &alpha, b, k, a, k, &beta_nosum, c, n, num_matr);
}

const void productoMatricesTrasposedABatchDevice(hipblasHandle_t handle, float** a, float** b, float** c, int m, int k, int n, int num_matr) {
    hipblasSgemmBatched_64(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, m, k, &alpha, b, n, a, m, &beta_nosum, c, n, num_matr);
}

const void productoMatricesBatchDeviceSumC(hipblasHandle_t handle, float** a, float** b, float** c, int m, int k, int n, int num_matr) {
    hipblasSgemmBatched_64(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, b, n, a, k, &beta_sum, c, n, num_matr);
}

__global__ void applyFunctionVectorial(float* arr, func_t func) {
    //https://forums.developer.nvidia.com/t/the-float-and-float4-types-in-cuda/65061
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float4 val = reinterpret_cast<float4*>(arr)[idx];
    val.x = func(val.x);
    val.y = func(val.y);
    val.z = func(val.z);
    val.w = func(val.w);
    reinterpret_cast<float4*>(arr)[idx] = val;
}

__global__ void applyFunctionScalar(float* arr, func_t func) {
    //https://forums.developer.nvidia.com/t/the-float-and-float4-types-in-cuda/65061
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    arr[idx] = func(arr[idx]);
}

__global__ void applyLossFunctionVectorial(float* pred, float* real, float* dst, func2_t func) {
    //https://forums.developer.nvidia.com/t/the-float-and-float4-types-in-cuda/65061
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float4 vpred = reinterpret_cast<float4*>(pred)[idx];
    float4 vreal = reinterpret_cast<float4*>(real)[idx];
    vpred.x = func(vpred.x, vreal.x);
    vpred.y = func(vpred.y, vreal.y);
    vpred.z = func(vpred.z, vreal.z);
    vpred.w = func(vpred.w, vreal.w);
    reinterpret_cast<float4*>(dst)[idx] = vpred;
}

__global__ void applyLossFunctionScalar(float* pred, float* real, float* dst, func2_t func) {
    //https://forums.developer.nvidia.com/t/the-float-and-float4-types-in-cuda/65061
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dst[idx] = func(pred[idx], real[idx]);
}

__global__ void multiplyAllElementsByConstantVectorial(float* arr, float ct) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float4 val = reinterpret_cast<float4*>(arr)[idx];
    val.x = val.x * ct;
    val.y = val.y * ct;
    val.z = val.z * ct;
    val.w = val.w * ct;
    reinterpret_cast<float4*>(arr)[idx] = val;
}

__global__ void multiplyAllElementsByConstantScalar(float* arr, float ct) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    arr[idx] = arr[idx] * ct;
}

__global__ void sumVectorsSameDimensionsVectorial(float* dst, float* src) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float4 val_src = reinterpret_cast<float4*>(src)[idx];
    float4 val_dst = reinterpret_cast<float4*>(dst)[idx];
    val_src.x = val_src.x + val_dst.x;
    val_src.y = val_src.y + val_dst.y;
    val_src.z = val_src.z + val_dst.z;
    val_src.w = val_src.w + val_dst.w;
    reinterpret_cast<float4*>(dst)[idx] = val_src;
}

__global__ void sumVectorsSameDimensionsScalar(float* dst, float* src) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dst[idx] = dst[idx] + src[idx];
}

__global__ void multiplyMatricesSameDimensionsVectorial(float* dst, float* src) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float4 val_src = reinterpret_cast<float4*>(src)[idx];
    float4 val_dst = reinterpret_cast<float4*>(dst)[idx];
    val_src.x = val_src.x * val_dst.x;
    val_src.y = val_src.y * val_dst.y;
    val_src.z = val_src.z * val_dst.z;
    val_src.w = val_src.w * val_dst.w;
    reinterpret_cast<float4*>(dst)[idx] = val_src;
}

__global__ void multiplyMatricesSameDimensionsScalar(float* dst, float* src) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dst[idx] = dst[idx] + src[idx];
}

const void managedApplyFunction(hipStream_t stream, int max_num_threads, int num_elems, float* arr, func_t func) {
    int nblocks = (int)(num_elems / (4 * max_num_threads));
    applyFunctionVectorial <<< nblocks, max_num_threads, 0, stream >> > (arr, func);
    num_elems -= (nblocks * 4 * max_num_threads);
    int offset = (nblocks * 4 * max_num_threads);
    applyFunctionVectorial << < 1, (int) (num_elems/4), 0, stream >> > (arr + offset, func);
    offset += ((num_elems / 4) * 4);
    num_elems -= (num_elems / 4) * 4;
    applyFunctionScalar << < 1, num_elems%4, 0, stream >> > (arr + offset, func);
}

const void managedApplyLossFunction(hipStream_t stream, int max_num_threads, int num_elems, float* pred, float* real, float* dst, func2_t func) {
    int nblocks = (int)(num_elems / (4 * max_num_threads));
    applyLossFunctionVectorial << < nblocks, max_num_threads, 0, stream >> > (pred, real, dst, func);
    num_elems -= (nblocks * 4 * max_num_threads);
    int offset = (nblocks * 4 * max_num_threads);
    applyLossFunctionVectorial << < 1, (int)(num_elems / 4), 0, stream >> > (pred+ offset, real+ offset, dst+ offset, func);
    offset += ((num_elems / 4) * 4);
    num_elems -= (num_elems / 4) * 4;
    applyLossFunctionScalar << < 1, num_elems % 4, 0, stream >> > (pred + offset, real + offset, dst + offset, func);
}

const void managedMultiplyAllElementsByConstant(hipStream_t stream, int max_num_threads, int num_elems, float* arr, float ct) {
    int nblocks = (int)(num_elems / (4 * max_num_threads));
    multiplyAllElementsByConstantVectorial << < nblocks, max_num_threads, 0, stream >> > (arr, ct);
    num_elems -= (nblocks * 4 * max_num_threads);
    int offset = (nblocks * 4 * max_num_threads);
    multiplyAllElementsByConstantVectorial << < 1, (int)(num_elems / 4), 0, stream >> > (arr + offset, ct);
    offset += ((num_elems / 4) * 4);
    num_elems -= (num_elems / 4) * 4;
    multiplyAllElementsByConstantScalar << < 1, num_elems % 4, 0, stream >> > (arr + offset, ct);
}

const void managedSumVectorsSameDimensions(hipStream_t stream, int max_num_threads, int num_elems, float* dst, float* src) {
    int nblocks = (int)(num_elems / (4 * max_num_threads));
    sumVectorsSameDimensionsVectorial << < nblocks, max_num_threads, 0, stream >> > (dst, src);
    num_elems -= (nblocks * 4 * max_num_threads);
    int offset = (nblocks * 4 * max_num_threads);
    sumVectorsSameDimensionsVectorial << < 1, (int)(num_elems / 4), 0, stream >> > (dst + offset, src + offset);
    offset += ((num_elems / 4) * 4);
    num_elems -= (num_elems / 4) * 4;
    sumVectorsSameDimensionsScalar << < 1, num_elems % 4, 0, stream >> > (dst + offset, src + offset);
}

const void managedMultiplyMatricesSameDimensions(hipStream_t stream, int max_num_threads, int num_elems, float* dst, float* src) {
    int nblocks = (int)(num_elems / (4 * max_num_threads));
    multiplyMatricesSameDimensionsVectorial << < nblocks, max_num_threads, 0, stream >> > (dst, src);
    num_elems -= (nblocks * 4 * max_num_threads);
    int offset = (nblocks * 4 * max_num_threads);
    multiplyMatricesSameDimensionsVectorial << < 1, (int)(num_elems / 4), 0, stream >> > (dst + offset, src + offset);
    offset += ((num_elems / 4) * 4);
    num_elems -= (num_elems / 4) * 4;
    multiplyMatricesSameDimensionsScalar << < 1, num_elems % 4, 0, stream >> > (dst + offset, src + offset);
}